#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t cu_add(int* a, int* b, int* c, int arr_size);
hipError_t cu_multi(float* c_m, float* a_m, float* b_m, int ha, int n, int wb);

__global__ void kernel_add(int* a, int* b, int* c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


__global__ void kernel_multi(float* c_m, float* a_m, float* b_m, int ha, int n, int wb) {
	int tid_x = threadIdx.x;
	int tid_y = threadIdx.y;
	//int bid_x = blockIdx.x;
	//int bid_y = blockIdx.y;
	float sum = 0;
	for (int i = 0; i != n; ++i) {
		//printf("(%f,%f,%d,%d,%d)\n", a_m[tid_y * n + i], b_m[i * n + tid_x],tid_x, tid_y, i);
		sum += a_m[tid_y * n + i] * b_m[i * wb + tid_x];
	}
	c_m[tid_y*wb+tid_x] = sum;
}

int main() {
	const int arraysize = 10;
	int a[arraysize] = { 0,1,2,3,4,5,6,7,8,9 };
	int b[arraysize] = { 9,8,7,6,5,4,3,2,1,0 };
	int c[arraysize] = { 0 };

	const int ha = 2, wb = 2, n = 3;
	/*
	float a_m[ha][n] = { {0,1,2},{3,4,5} };
	float b_m[n][wb] = { {2,3},{4,5},{6,7} };
	float c_m[ha][wb] = { 0 };
	*/

	float a_m[ha * n] = { 0,1,2,3,4,5 };
	float b_m[n * wb] = { 2,3,4,5,6,7 };
	float c_m[ha * wb] = { 0 };
	

	hipError_t cu_stat;

	cu_stat = cu_add(a, b, c, arraysize);
	if (cu_stat != hipSuccess) {
		cout << "error in cu_add" << endl;
		return 1;
	}

	cout << "the result is: " << c[0]<< c[1]<< c[2]<< c[3] << endl;

	cu_stat = cu_multi(c_m, a_m, b_m, ha, n, wb);
	if (cu_stat != hipSuccess) {
		cout << "error in cu_add" << endl;
		return 1;
	}

	for (int i = 0; i != ha; ++i) {
		for (int j = 0; j != wb; ++j) {
			cout << c_m[i * wb + j] << " ";
		}
		cout << endl;
	}

	hipDeviceReset();


	return 0;



}


hipError_t cu_add(int* a, int* b, int* c, int arr_size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	hipError_t cuda_stat;
	cuda_stat = hipSetDevice(0);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	cout << "device name: " << deviceProp.name << endl;
	cout << "the warpsize: " << deviceProp.warpSize << endl;
	cout << "the maxThreadPerBlock: " << deviceProp.maxThreadsPerBlock << endl;
	cout << "the maxThreadsDim: " << deviceProp.maxThreadsDim[0] << endl;
	cout << "the maxGridSize: " << deviceProp.maxGridSize[0] << endl;
	cout << "the compute capability: " << deviceProp.major << '.' << deviceProp.minor << endl;

	if (cuda_stat != hipSuccess) {
		cout << "error in hipSetDevice" << endl;
		goto Error;
	}

	hipMalloc((void**)&dev_a, arr_size * sizeof(int));
	hipMalloc((void**)&dev_b, arr_size * sizeof(int));
	hipMalloc((void**)&dev_c, arr_size * sizeof(int));

	cuda_stat = hipMemcpy(dev_a, a, arr_size * sizeof(int), hipMemcpyHostToDevice);
	cuda_stat = hipMemcpy(dev_b, b, arr_size * sizeof(int), hipMemcpyHostToDevice);
	cuda_stat = hipMemcpy(dev_c, c, arr_size * sizeof(int), hipMemcpyHostToDevice);

	kernel_add << <1, arr_size >> > (dev_a, dev_b, dev_c);

	cuda_stat = hipGetLastError();
	if (cuda_stat != hipSuccess) {
		cout << "error in calling the kernel_add" << endl;
		goto Error;
	}

	cuda_stat = hipDeviceSynchronize();
	cuda_stat = hipMemcpy(c, dev_c, arr_size * sizeof(int), hipMemcpyDeviceToHost);
	if (cuda_stat != hipSuccess) {
		cout << "error in copy mem from D to H" << endl;
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return cuda_stat;
}

hipError_t cu_multi(float* c_m, float* a_m, float* b_m, int ha, int n, int wb) {
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;

	dim3 threads(ha, wb);

	hipError_t cuda_stat;
	cuda_stat = hipSetDevice(0);
	if (cuda_stat != hipSuccess) {
		cout << "error in setting device" << endl;
		goto Error;
	}

	hipMalloc((void**)&dev_a, ha * n * sizeof(float));
	hipMalloc((void**)&dev_b, n * wb * sizeof(float));
	hipMalloc((void**)&dev_c, ha * wb * sizeof(float));

	hipMemcpy(dev_a, a_m, ha * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b_m, n * wb * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c_m, ha * wb * sizeof(float), hipMemcpyHostToDevice);

	kernel_multi << <1, threads >> > (dev_c, dev_a, dev_b, ha, n, wb);
	cuda_stat = hipGetLastError();
	if (cuda_stat != hipSuccess) {
		cout << "error in calling kernel_multi" << endl;
		goto Error;
	}

	cuda_stat = hipDeviceSynchronize();
	cuda_stat = hipMemcpy(c_m, dev_c, ha * wb * sizeof(float), hipMemcpyDeviceToHost);
	if (cuda_stat != hipSuccess) {
		cout << "error in copy from D to H";
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return cuda_stat;
}